#include "hip/hip_runtime.h"
// Copyright (c) 2022 Fetullah Atas, Norwegian University of Life Sciences
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "vox_nav_cupoch/cupoch_gpu_icp.hpp"

#include <string>
#include <vector>
#include <memory>
#include <algorithm>

using namespace vox_nav_cupoch;

CupochGPUICP::CupochGPUICP()
    : Node("cupoch_gpu_icp_rclcpp_node")
{

    live_cloud_subscriber_ = this->create_subscription<sensor_msgs::msg::PointCloud2>(
        "/ouster/points",
        rclcpp::SensorDataQoS(),
        std::bind(
            &CupochGPUICP::liveCloudCallback, this, std::placeholders::_1));

    map_cloud_subscriber_ = this->create_subscription<sensor_msgs::msg::PointCloud2>(
        "vox_nav/map_server/octomap_pointcloud",
        rclcpp::SensorDataQoS(),
        std::bind(
            &CupochGPUICP::mapCloudCallback, this, std::placeholders::_1));

    gps_odom_subscriber_ = this->create_subscription<nav_msgs::msg::Odometry>(
        "odometry/gps",
        rclcpp::SensorDataQoS(),
        std::bind(&CupochGPUICP::gpsOdomCallback, this, std::placeholders::_1));

    live_cloud_pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
        "vox_nav/cupoch/live_cloud_crop", rclcpp::SystemDefaultsQoS());

    map_cloud_pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
        "vox_nav/cupoch/map_cloud_crop", rclcpp::SystemDefaultsQoS());

    base_to_map_pose_pub_ = this->create_publisher<geometry_msgs::msg::PoseWithCovarianceStamped>(
        "vox_nav/cupoch/icp_base_to_map_pose", rclcpp::SystemDefaultsQoS());

    new_robot_pose_publisher_ =
        this->create_publisher<geometry_msgs::msg::PoseArray>(
            "vox_nav/cupoch/icp_robot_pose", rclcpp::SystemDefaultsQoS());

    // setup TF buffer and listerner to read transforms
    tf_buffer_ = std::make_shared<tf2_ros::Buffer>(this->get_clock());
    tf_listener_ = std::make_shared<tf2_ros::TransformListener>(*tf_buffer_);

    latest_gps_odom_ = std::make_shared<nav_msgs::msg::Odometry>();
    map_cloud_ = pcl::make_shared<pcl::PointCloud<pcl::PointXYZRGB>>();

    // Define parameters
    declare_parameter("x_bound", params_.x_bound);
    declare_parameter("y_bound", params_.y_bound);
    declare_parameter("z_bound", params_.z_bound);
    declare_parameter("downsample_voxel_size", params_.downsample_voxel_size);
    declare_parameter("max_icp_iter", params_.max_icp_iter);
    declare_parameter("max_correspondence_distance", params_.max_correspondence_distance);
    declare_parameter("debug", params_.debug);

    get_parameter("x_bound", params_.x_bound);
    get_parameter("y_bound", params_.y_bound);
    get_parameter("z_bound", params_.z_bound);
    get_parameter("downsample_voxel_size", params_.downsample_voxel_size);
    get_parameter("max_icp_iter", params_.max_icp_iter);
    get_parameter("max_correspondence_distance", params_.max_correspondence_distance);
    get_parameter("debug", params_.debug);

    // Print parameters
    RCLCPP_INFO_STREAM(get_logger(), "x_bound " << params_.x_bound);
    RCLCPP_INFO_STREAM(get_logger(), "y_bound " << params_.y_bound);
    RCLCPP_INFO_STREAM(get_logger(), "z_bound " << params_.z_bound);
    RCLCPP_INFO_STREAM(get_logger(), "downsample_voxel_size " << params_.downsample_voxel_size);
    RCLCPP_INFO_STREAM(get_logger(), "max_icp_iter " << params_.max_icp_iter);
    RCLCPP_INFO_STREAM(
        get_logger(), "max_correspondence_distance " << params_.max_correspondence_distance);
    RCLCPP_INFO_STREAM(get_logger(), "debug " << params_.debug);

    RCLCPP_INFO(get_logger(), "Creating...");
}

CupochGPUICP::~CupochGPUICP()
{
    RCLCPP_INFO(get_logger(), "Destroying...");
}

void CupochGPUICP::gpsOdomCallback(
    const nav_msgs::msg::Odometry::ConstSharedPtr odom)
{
    std::lock_guard<std::mutex> guard(latest_gps_odom_mutex_);
    latest_gps_odom_ = std::make_shared<nav_msgs::msg::Odometry>(*odom);
}

void CupochGPUICP::liveCloudCallback(
    const sensor_msgs::msg::PointCloud2::ConstSharedPtr cloud)
{
    if (map_configured_)
    {
        std::lock_guard<std::mutex> guard(latest_gps_odom_mutex_);

        pcl::PointCloud<pcl::PointXYZRGB>::Ptr pcl_curr(new pcl::PointCloud<pcl::PointXYZRGB>());
        pcl::fromROSMsg(*cloud, *pcl_curr);

        if (!pcl_ros::transformPointCloud(
                "base_link", *pcl_curr, *pcl_curr, *tf_buffer_))
        {
            RCLCPP_WARN(get_logger(), "Error Encountered at transfrom, doing nothing");
            latest_icp_pose_.header.stamp = now();
            base_to_map_pose_pub_->publish(latest_icp_pose_);
            return;
        }

        auto croppped_live_cloud = vox_nav_utilities::cropBox<pcl::PointXYZRGB>(
            pcl_curr,
            Eigen::Vector4f(-params_.x_bound, -params_.y_bound, -params_.z_bound, 1),
            Eigen::Vector4f(params_.x_bound, params_.y_bound, params_.z_bound, 1));

        geometry_msgs::msg::PoseStamped curr_robot_pose;

        curr_robot_pose.header.frame_id = "map";
        curr_robot_pose.header.stamp = cloud->header.stamp;
        curr_robot_pose.pose = latest_gps_odom_->pose.pose;

        auto croppped_map_cloud = vox_nav_utilities::cropBox<pcl::PointXYZRGB>(
            map_cloud_,
            Eigen::Vector4f(
                -params_.x_bound + curr_robot_pose.pose.position.x,
                -params_.y_bound + curr_robot_pose.pose.position.y,
                -params_.z_bound + curr_robot_pose.pose.position.z, 1),

            Eigen::Vector4f(
                params_.x_bound + curr_robot_pose.pose.position.x,
                params_.y_bound + curr_robot_pose.pose.position.y,
                params_.z_bound + curr_robot_pose.pose.position.z, 1));

        croppped_map_cloud->header.frame_id = "map";
        croppped_map_cloud->header.stamp = pcl_curr->header.stamp;
        croppped_map_cloud->header.seq = pcl_curr->header.seq;

        if (!pcl_ros::transformPointCloud(
                "base_link", *croppped_map_cloud, *croppped_map_cloud, *tf_buffer_))
        {
            RCLCPP_WARN(get_logger(), "Error Encountered at transfrom, doing nothing");
            latest_icp_pose_.header.stamp = now();
            base_to_map_pose_pub_->publish(latest_icp_pose_);
            return;
        }

        croppped_map_cloud =
            vox_nav_utilities::downsampleInputCloud<pcl::PointXYZRGB>(
                croppped_map_cloud, params_.downsample_voxel_size);
        croppped_live_cloud =
            vox_nav_utilities::downsampleInputCloud<pcl::PointXYZRGB>(
                croppped_live_cloud, params_.downsample_voxel_size);

        thrust::host_vector<Eigen::Vector3f> map_points, live_points;

        for (int i = 0; i < croppped_map_cloud->points.size(); ++i)
        {
            auto p = croppped_map_cloud->points[i];
            Eigen::Vector3f point_eig(p.x, p.y, p.z);
            map_points.push_back(point_eig);
        }

        for (int i = 0; i < croppped_live_cloud->points.size(); ++i)
        {
            auto p = croppped_live_cloud->points[i];
            Eigen::Vector3f point_eig(p.x, p.y, p.z);
            live_points.push_back(point_eig);
        }

        auto map_points_cupoch = std::make_shared<cupoch::geometry::PointCloud>();
        auto live_points_cupoch = std::make_shared<cupoch::geometry::PointCloud>();
        map_points_cupoch->SetPoints(map_points);
        live_points_cupoch->SetPoints(live_points);

        // ICP
        auto point_to_point =
            cupoch::registration::TransformationEstimationPointToPoint();
        cupoch::registration::ICPConvergenceCriteria criteria;
        criteria.max_iteration_ = params_.max_icp_iter;
        auto res = cupoch::registration::RegistrationICP(
            *live_points_cupoch, *map_points_cupoch, params_.max_correspondence_distance,
            Eigen::Matrix4f::Identity(), point_to_point, criteria);

        if (std::abs(res.transformation_(0, 3)) > 0.5 ||
            std::abs(res.transformation_(1, 3)) > 0.5 ||
            std::abs(res.transformation_(2, 3)) > 0.5)
        {

            res.transformation_(0, 3) = clamp<double>(res.transformation_(0, 3), -0.5, 0.5);
            res.transformation_(1, 3) = clamp<double>(res.transformation_(1, 3), -0.5, 0.5);
            res.transformation_(2, 3) = clamp<double>(res.transformation_(2, 3), -0.5, 0.5);
        }

        live_points_cupoch->Transform(res.transformation_);

        Eigen::Affine3f T;
        T.matrix() = res.transformation_;
        Eigen::Affine3d T_d = T.cast<double>();
        auto transformation = tf2::eigenToTransform(T_d);

        geometry_msgs::msg::PoseStamped a, b;
        tf2::doTransform(a, b, transformation);

        b.header.frame_id = "base_link";
        b.header.stamp = curr_robot_pose.header.stamp;
        a.header.stamp = curr_robot_pose.header.stamp;

        rclcpp::Duration transform_tolerance(0, 500);

        auto result = vox_nav_utilities::transformPose(
            tf_buffer_, "map", b, a, transform_tolerance);

        geometry_msgs::msg::PoseArray icp_robot_poses;
        icp_robot_poses.header.frame_id = "map";
        icp_robot_poses.header.stamp = now();
        icp_robot_poses.poses.push_back(curr_robot_pose.pose);
        icp_robot_poses.poses.push_back(a.pose);
        new_robot_pose_publisher_->publish(icp_robot_poses);

        geometry_msgs::msg::PoseWithCovarianceStamped icp_pose;
        icp_pose.header.stamp = curr_robot_pose.header.stamp;
        icp_pose.header.frame_id = "map";
        icp_pose.pose.pose = a.pose;
        latest_icp_pose_ = icp_pose;
        base_to_map_pose_pub_->publish(icp_pose);

        pcl::PointCloud<pcl::PointXYZRGB>::Ptr live_cloud_crop(new pcl::PointCloud<pcl::PointXYZRGB>());
        pcl::PointCloud<pcl::PointXYZRGB>::Ptr map_cloud_crop(new pcl::PointCloud<pcl::PointXYZRGB>());

        for (auto &&i : live_points_cupoch->GetPoints())
        {
            pcl::PointXYZRGB p;
            p.x = i.x();
            p.y = i.y();
            p.z = i.z();
            p.r = 255;
            p.a = 200;
            live_cloud_crop->points.push_back(p);
        }

        for (auto &&i : map_points_cupoch->GetPoints())
        {
            pcl::PointXYZRGB p;
            p.x = i.x();
            p.y = i.y();
            p.z = i.z();
            p.b = 255;
            p.a = 200;
            map_cloud_crop->points.push_back(p);
        }

        sensor_msgs::msg::PointCloud2 live_cloud_crop_msg, map_cloud_crop_msg;

        pcl::toROSMsg(*live_cloud_crop, live_cloud_crop_msg);
        pcl::toROSMsg(*map_cloud_crop, map_cloud_crop_msg);
        live_cloud_crop_msg.header = cloud->header;
        live_cloud_crop_msg.header.frame_id = "base_link";
        map_cloud_crop_msg.header = cloud->header;
        map_cloud_crop_msg.header.frame_id = "base_link";

        live_cloud_pub_->publish(live_cloud_crop_msg);
        map_cloud_pub_->publish(map_cloud_crop_msg);

        last_transform_estimate_ = res.transformation_;

        if (params_.debug)
        {
            RCLCPP_INFO(
                get_logger(), "Did ICP with Live Cloud of %d points...", live_cloud_crop->points.size());
            RCLCPP_INFO(
                get_logger(), "Did ICP with Map Cloud of %d points...", map_cloud_crop->points.size());
            std::cout << "Resulting transfrom: \n"
                      << res.transformation_ << std::endl;
        }
    }
}

void CupochGPUICP::mapCloudCallback(
    const sensor_msgs::msg::PointCloud2::ConstSharedPtr cloud)
{
    std::call_once(
        get_map_cloud_once_, [&]()
        {
      pcl::fromROSMsg(*cloud, *map_cloud_);
      map_configured_ = true;
      RCLCPP_INFO(get_logger(), "Map Cloud with %d points...", map_cloud_->points.size()); });
}

int main(int argc, char const *argv[])
{
    rclcpp::init(argc, argv);
    auto node = std::make_shared<CupochGPUICP>();
    rclcpp::spin(node);
    rclcpp::shutdown();
    return 0;
}
